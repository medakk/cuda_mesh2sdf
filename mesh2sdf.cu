
#include <hip/hip_runtime.h>
// centroid:  [  92.6200991  -157.6624484  -666.61104378]
// scale:  [1.38349843 0.99729681 2.00067234


inline __host__ __device__ float3 operator-(float3 a, float3 b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

inline __host__ __device__ float3 cross(float3 a, float3 b)
{
    return make_float3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x);
}

inline __host__ __device__ float dot(float3 a, float3 b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

inline __host__ __device__ float dot2(float3 a)
{
    return dot(a,a);
}

inline __device__ __host__ float clamp(float f, float a, float b)
{
    return fmaxf(a, fminf(f, b));
}
inline __host__ __device__ float3 operator*(float3 a, float b)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}
inline __host__ __device__ float sign(float x)
{
    float t = x > 0.0;
    return t - (x < 0.0);
}



// inigo quilez: https://www.iquilezles.org/www/articles/triangledistance/triangledistance.htm
__device__
float distance( float3 v1, float3 v2, float3 v3, float3 p )
{
    // prepare data    
    float3 v21 = v2 - v1; float3 p1 = p - v1;
    float3 v32 = v3 - v2; float3 p2 = p - v2;
    float3 v13 = v1 - v3; float3 p3 = p - v3;
    float3 nor = cross( v21, v13 );

    float dist = sqrt( // inside/outside test    
                 ( sign(dot(cross(v21,nor),p1))
                 + sign(dot(cross(v32,nor),p2))
                 + sign(dot(cross(v13,nor),p3))<2.0) 
                  ?
                  // 3 edges    
                  min( min( 
                  dot2(v21*clamp(dot(v21,p1)/dot2(v21),0.0,1.0)-p1), 
                  dot2(v32*clamp(dot(v32,p2)/dot2(v32),0.0,1.0)-p2) ), 
                  dot2(v13*clamp(dot(v13,p3)/dot2(v13),0.0,1.0)-p3) )
                  :
                  // 1 face    
                  dot(nor,p1)*dot(nor,p1)/dot2(nor) );
    
    // which side of the triangle?
    return sign(dot(nor, p1)) * dist;
}


__global__ void mesh2sdf(float *sdf, int w, int h, int d, float *V, int *F, int nFaces)
{
    const uint y = (blockIdx.y * blockDim.y) + threadIdx.y;
    const uint z = (blockIdx.z * blockDim.z) + threadIdx.z;

    // TODO is this right? (most definitely not)
    if(y >= h || z >= d) {
        return;
    }

    // todo pass in scale dont hardcode 
    const float pt_y = (y - h / 2.0) * 0.99729681 * 64.0 / (float) h;
    const float pt_z = (z - d / 2.0) * 2.00067234 * 64.0 / (float) d;

    for(uint x=0; x<w; x++) {
        const int idx = x + w * (y + d * z);
        float currDist = sdf[idx];

        const float pt_x = (x - w / 2.0) * 1.38349843 * 64.0 / (float) w;
        float3 pt = make_float3(pt_x, pt_y, pt_z);

        for(int f=0; f<nFaces; f++) {
            float3 v1 = make_float3(V[3*F[3*f+0]+0], V[3*F[3*f+0]+1], V[3*F[3*f+0]+2]);
            float3 v2 = make_float3(V[3*F[3*f+1]+0], V[3*F[3*f+1]+1], V[3*F[3*f+1]+2]);
            float3 v3 = make_float3(V[3*F[3*f+2]+0], V[3*F[3*f+2]+1], V[3*F[3*f+2]+2]);

            const float dist = distance(v1, v2, v3, pt);

            if(abs(dist) < abs(currDist)) {
                currDist = dist;
            }
        }

        sdf[idx] = currDist;
    }
}